#include "hip/hip_runtime.h"
//
//  techniques.cpp
//  Coordinate_descent
//
//  Created by Zhiwei Fan on 10/13/15.
//  Edited by Huawei Wang on 10/03/18, new CUDA 5.0 version
//  Copyright © 2015 Zhiwei Fan. All rights reserved.
//

#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <time.h>
#include "techniques.h"
#include "DataManagement.h"
#include "linear_models.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include "math.h"
techniques::techniques(){};

/*
lm becomes class variable, calls device function like below
//Compute the partial gradient
// if to avoid overhead, may be here not much dimension of this single model, just update paramas in the host
    cudaMemcopy(dmodel, model + j, 1*sizeof(double), hipMemcpyHostToDevice);
*/

__global__ void gradientkl(double* dY, double* dH, double* dX, double* dmul_arr, long row_num) {
    int Idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(Idx < row_num) {
        dmul_arr[Idx] =  -dY[Idx]/(1+exp(dY[Idx]*dH[Idx]))*dX[Idx];
    }
    
}

__global__ void backkl(double* dH, double* dX, double diff, long row_num) {
	int Idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(Idx < row_num) {
        // dH[Idx] = dH[Idx] + diff * dX[Idx];
        dH[Idx] = __fma_rn(diff, dX[Idx], dH[Idx]);
    }
}

__global__ void losskl(double* dY, double* dH, double* dFcur, long row_num) {
	int Idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(Idx < row_num) {
        // dFcur[Idx] = log1pf(-expf(dY[Idx]*dH[Idx]));
        dFcur[Idx] = log(1 + exp(-dY[Idx]*dH[Idx]));
    }
}
/**
 Coordinate Descent/Block Coordinate Descent:
 (CD/BCD)
 Materialize, Stream, Factorize
 
 Stochastic Gradient Descent/Batch Gradient Descent:
 (SGD/BGD)
 Materialize only

 **/
//Just Logistic Regression by now
__host__ void techniques::materialize(string table_T, setting _setting, double *model)
{
    DataManagement DM;
    DM.message("Start materialize");
    linear_models lm;
    vector<long> tableInfo(3);
    vector<string> fields = DM.getFieldNames(table_T, tableInfo);
    int feature_num = (int)tableInfo[1];
    long row_num = 100000; //tableInfo[2];
    
    double *Y;
    double *H;
    double *X;
    double *mul_arr;

    hipEvent_t startEvent_exc, stopEvent_exc;
	hipEventCreate(&startEvent_exc);
	hipEventCreate(&stopEvent_exc);
	float elapsedTime_exc;

    //setting
    double step_size = _setting.step_size;
    
    //Allocate the memory to the model
    model = new double[feature_num];
    //Allocate the memory to the label Array
    Y = new double[row_num];
    //Allocate the memory to H
    H = new double[row_num];
    //Allocate the memory to X
    X = new double[row_num];
    // Temp variable for recording the multiply value
	mul_arr = new double[row_num];
    double *dY;
    double *dH;
    double *dX;
    double *dmul_arr;
    double *dFcur;
    // double *dmodel;
	// Allocate Device variables need for computing
    hipMalloc((void**)&dY, row_num*sizeof(double));
    hipMalloc((void**)&dH, row_num*sizeof(double));
    hipMalloc((void**)&dX, row_num*sizeof(double)); 
    hipMalloc((void**)&dmul_arr, row_num*sizeof(double));   //临时变量          
    // hipMalloc((void**)&dmodel, 1*sizeof(double));           //计算变量                        
    hipMalloc((void**)&dFcur, row_num*sizeof(double));
    hipMemset(dH, 0, row_num*sizeof(double));
	// kernel parameter
    const int threadsPerBlock = 1024;
    const int blocksPerGrid = row_num/threadsPerBlock + 1;
    
    double F = 0.00;
    double F_partial = 0.00;
    double r_curr = 0.00;
    double r_prev = 0.00;
    int k = 0;
    
    
    for(int i = 0; i < feature_num; i ++)
    {
        model[i] = 0.00;
    }
   
	// Fetch the label column 1 time, leave it on GPU memory
	// May move to shared memory to improve efficiency
 
    DM.fetchColumn(fields[1], row_num, Y);
    hipMemcpy(dY, Y, row_num*sizeof(double), hipMemcpyHostToDevice);
    //First do Logistic Regression
    do
    {
        //Update one coordinate each time
        for(int j = 0; j < feature_num; j ++)
        {
            F_partial = 0.00;
            //Fetch the each column and store the current column into X
            DM.fetchColumn(fields[2+j], row_num, X);
	        
            hipEventRecord(startEvent_exc,0); // staring timing for exclusive
            hipMemcpy(dX, X, row_num*sizeof(double), hipMemcpyHostToDevice);
            // launch the kernal only 1 time
            hipEventRecord(stopEvent_exc,0);  // ending timing for exclusive
            hipEventSynchronize(stopEvent_exc);   
            hipEventElapsedTime(&elapsedTime_exc, startEvent_exc, stopEvent_exc);
            cout << "Gradient recuce time " << elapsedTime_exc << endl;

            gradientkl<<<blocksPerGrid, threadsPerBlock>>>(dY, dH, dX, dmul_arr, row_num);
			hipDeviceSynchronize();

            // Reduce to get the sum of current gradient
			hipMemcpy(mul_arr, dmul_arr, row_num*sizeof(double), hipMemcpyDeviceToHost);
			F_partial = thrust::reduce(thrust::device, dmul_arr, dmul_arr + row_num, 0.0);

            // for(long i = 0; i < row_num; i++) {
            //     F_partial += mul_arr[i];
            // }

            double W_j = model[j];
            //Update the current coordinate
            model[j]  = model[j] - step_size * F_partial;
            double diff = model[j] - W_j;
            
            backkl<<<blocksPerGrid, threadsPerBlock>>>(dH, dX, diff, row_num);
            hipDeviceSynchronize();
        }
        hipMemcpy(H, dH, row_num*sizeof(double), hipMemcpyDeviceToHost);										
        r_prev = F;
        //Caculate F
        F = 0.00;
        /*
        for(long i = 0; i < row_num ; i ++)
        {
            double tmp = lm.Fe_lr(Y[i],H[i]);
            
            if(i < 5){
                cout<<"Yi: "<<Y[i]<<","<<"Hi: "<<H[i]<<","<<"tmp: "<<tmp<<endl;
            }
            
            
            F += tmp;
        }
        */
        
        losskl<<<blocksPerGrid, threadsPerBlock>>>(dY, dH, dFcur, row_num);
        
        F = thrust::reduce(thrust::device, dFcur, dFcur + row_num, 0.0);
        r_curr = F;

        k++;
        cout<<"Model: "<<endl;
        
        for(int i = 0; i < 10; i ++)
        {
            cout<<model[i]<< " ";
        }
        
        //cout<<endl;
        cout<<"Loss value: "<<F<<endl;
        
    }
    while(!stop(k,r_prev,r_curr,_setting));
    hipFree(dY);
    hipFree(dH);
    hipFree(dX);
    hipFree(dmul_arr);
    // hipFree(dmodel);
    delete [] Y;
    delete [] X;
    delete [] H;
    
    printf("The final loss: %lf\n",r_curr);
    printf("Number of iteration: %d\n",k);
    printf("Model: ");
    for(int i = 0; i < feature_num; i ++)
    {
        if(i == feature_num - 1)
        {
            printf("%.20f\n",model[i]);
        }
        else
        {
             printf("%.20f, ",model[i]);
        }
       
    }
    
    DM.message("Finish materialize");
    
}

/* Should be no oid-oid mapping here */
__host__ void techniques::stream(string table_S, string table_R, setting _setting, double *model)
{
    DataManagement DM;
    DM.message("Start stream");
    // linear_models lm; set as class variable
    
    //Get the table information and column names
    vector<long> tableInfo_S(3);
    vector<long> tableInfo_R(3);
    vector<string> fields_S = DM.getFieldNames(table_S, tableInfo_S);
    vector<string> fields_R = DM.getFieldNames(table_R, tableInfo_R);
    int feature_num_S = (int)tableInfo_S[1];
    int feature_num_R = (int)tableInfo_R[1];
    int feature_num = feature_num_S + feature_num_R;
    long row_num = tableInfo_S[2];
    long row_num_R = tableInfo_R[2];
    
    double *Y;
    double *H;
    double *X;
    
    //setting
    double step_size = _setting.step_size;
    
    //Allocate the memory to the model
    model = new double[feature_num_S + feature_num_R];
    //Allocate the memory to the label Array
    Y = new double[row_num];
    //Allocate the memory to H
    H = new double[row_num];
    //Allocate the memory to X
    X = new double[row_num];
    
    double F = 0.00;
    double F_partial = 0.00;
    double r_curr = 0.00;
    double r_prev = 0.00;
    int k = 0;
    
    for(int i = 0; i < feature_num; i ++)
    {
        model[i] = 0.00;
        
    }
    
    for(int i = 0; i < row_num; i ++)
    {
        H[i] = 0.00;
    }
    
    DM.fetchColumn(fields_S[1], row_num, Y);
    
    //First do Logistic Regression
    do
    {
        printf("Start fetching KKMR reference\n");
        //OID-OID Mapping (Key Foreign-Key Mapping Reference)
        double *KKMR = new double[row_num];
        //Read the fk column(referred rid in R) in table S, rid column in R
        ifstream fk;
        //Load the fk to KKMR
        fk.open(fields_S[2], ios::in | ios::binary);
        //rid.open(table2_fields[0], ios::in | ios::binary);
        if(!fk.is_open())
        {
            cerr<<"Error Message: "<<"Cannot load the fk column."<<endl;
            exit(1);
        }
        fk.read((char *)KKMR, row_num*(sizeof(double)));
        fk.close();
        printf("Finish fetchig KKMR reference\n");
        
        //Update one coordinate each time
        for(int j = 0; j < feature_num; j ++)
        {
            F_partial = 0.00;
            //Fetch each column in table S and store the current column into X
            if(j < feature_num_S)
            {
                DM.fetchColumn(fields_S[3+j], row_num, X);
            }
            else
            {
                double *tmp_buffer = new double[row_num_R];
                DM.fetchColumn(fields_R[1+j-feature_num_S], row_num_R, tmp_buffer);
                for(long m = 0; m < row_num; m ++)
                {
                    long fk = KKMR[m];
                    X[m]= tmp_buffer[fk-1];
                }
                delete tmp_buffer;
            }
            
//////////////////////These part the same as materialize before may just consider wrapping into a single function
            //Compute the partial gradient
            for(long i = 0; i < row_num ; i ++)
            {
                F_partial += lm.G_lr(Y[i],H[i])*X[i];
            }
            
            //Store the old W(j)
            double W_j = model[j];
            
            //Update the current coordinate
            model[j] = model[j] - step_size * F_partial;
            
            
            double diff = model[j] - W_j;
            
            //Update the intermediate variable
            //H = H + (Wj - old_Wj)* X(,j)
            for(long m = 0; m < row_num; m ++ )
            {
                H[m] = H[m] + diff*X[m];
            }
        }
        
        r_prev = F;
        //Caculate F
        F = 0.00;
        for(long i = 0; i < row_num ; i ++)
        {
            double tmp = lm.Fe_lr(Y[i],H[i]);
            F += tmp;
        }
        
        r_curr = F;
        k++;
        
        delete [] KKMR;
    } while(!stop(k,r_prev,r_curr,_setting));
    
    delete [] Y;
    delete [] X;
    delete [] H;
    
    
    printf("The final loss: %lf\n",r_curr);
    printf("Number of iteration: %d\n",k);
    printf("Model: ");
    for(int i = 0; i < feature_num; i ++)
    {
        if(i == feature_num - 1)
        {
            printf("%.20f\n",model[i]);
        }
        else
        {
            printf("%.20f, ",model[i]);
        }
        
    }
    
    DM.message("Finish stream");
}


__host__ void techniques::factorize(string table_S, string table_R, setting _setting, double *model)
{
    DataManagement DM;
    DM.message("Start factorize");
    linear_models lm;
    
    //Get the table information and column names
    vector<long> tableInfo_S(3);
    vector<long> tableInfo_R(3);
    vector<string> fields_S = DM.getFieldNames(table_S, tableInfo_S);
    vector<string> fields_R = DM.getFieldNames(table_R, tableInfo_R);
    int feature_num_S = (int)tableInfo_S[1];
    int feature_num_R = (int)tableInfo_R[1];
    int feature_num = feature_num_S + feature_num_R;
    long row_num = tableInfo_S[2];
    long row_num_R = tableInfo_R[2];
    
    printf("Start fetching KKMR reference\n");
    //OID-OID Mapping (Key Foreign-Key Mapping Reference)
    double *KKMR = new double[row_num];
    //Read the fk column(referred rid in R) in table S, rid column in R
    ifstream fk;
    //Load the fk to KKMR
    fk.open(fields_S[2], ios::in | ios::binary);
    //rid.open(table2_fields[0], ios::in | ios::binary);
    if(!fk.is_open())
    {
        cerr<<"Error Message: "<<"Cannot load the fk column."<<endl;
        exit(1);
    }
    fk.read((char *)KKMR, row_num*(sizeof(double)));
    fk.close();
    //printf("Finish fetchig KKMR reference\n");
    
    double *Y;
    double *H;
    double *X;
    
    //setting
    double step_size = _setting.step_size;
    
    //Allocate the memory to the model
    model = new double[feature_num_S + feature_num_R];
    //Allocate the memory to the label Array
    Y = new double[row_num];
    //Allocate the memory to H
    H = new double[row_num];
    //Allocate the memory to X
    X = new double[row_num];
    
    double F = 0.00;
    double F_partial = 0.00;
    double r_curr = 0.00;
    double r_prev = 0.00;
    int k = 0;
    
    for(int i = 0; i < feature_num; i ++)
    {
        model[i] = 0.00;
        
    }
    
    for(int i = 0; i < row_num; i ++)
    {
        H[i] = 0.00;
    }
    
    DM.fetchColumn(fields_S[1], row_num, Y);
    
    //First do Logistic Regression
    do
    {
        //Update one coordinate each time
        for(int j = 0; j < feature_num; j ++)
        {
            F_partial = 0.00;
            //Fetch each column in table S and store the current column into X
            if(j < feature_num_S)
            {
                DM.fetchColumn(fields_S[3+j], row_num, X);
//////////////////////////////////////////////////////Same part as before
                //Compute the partial gradient
                for(int i = 0; i < row_num ; i ++)
                {
                    F_partial += lm.G_lr(Y[i],H[i])*X[i];
                }
                
                //Store the old W(j)
                double W_j = model[j];
                
                //Update the current coordinate
                model[j] = model[j] - step_size * F_partial;
                cout<<"model("<<j<<"): "<<model[j]<<endl;
                
                double diff = model[j] - W_j;
                //Update the intermediate variable
                //H = H + (Wj - old_Wj)* X(,j)
                for(long m = 0; m < row_num; m ++ )
                {
                    H[m] = H[m] + diff*X[m];
                }
            }
            else
            {
                double **tmp_buffer = new double*[2];
                for(int i = 0; i < 2; i ++)
                {
                    tmp_buffer[i] = new double[row_num_R];
                    for(long k = 0; k < row_num_R; k ++)
                    {
                        tmp_buffer[i][k] = 0.00;
                    }
                }
                
                DM.fetchColumn(fields_R[1+j-feature_num_S], row_num_R, tmp_buffer[0]);
                //Compute the partial gradient
                for(long m = 0; m < row_num; m ++)
                {
                    long fk = KKMR[m];
                    tmp_buffer[1][fk-1] += lm.G_lr(Y[m],H[m]);
                }
                for(long j = 0; j < row_num_R; j ++)
                {
                    F_partial += tmp_buffer[0][j]*tmp_buffer[1][j];
                }
                
                
                //Store the old W(j)
                double W_j = model[j];
                
                //Update the current coordinate
                model[j] = model[j] - step_size * F_partial;
                cout<<"model("<<j<<"): "<<model[j]<<endl;
                
                double diff = model[j] - W_j;

                
                //Factorized computation
                for(long k = 0; k < row_num_R; k ++)
                {
                    tmp_buffer[0][k] = diff*tmp_buffer[0][k];
                }
                
                //Update the intermediate variable
                //H = H + (Wj - old_Wj)* X(,j)
                for(long m = 0; m < row_num; m ++ )
                {
                    long fk = KKMR[m];
                    H[m] = H[m] + tmp_buffer[0][fk-1];
                }

                
                delete [] tmp_buffer[0];
                delete [] tmp_buffer[1];
                delete [] tmp_buffer;
            }
            
        }
        
        r_prev = F;
        //Caculate F
        F = 0.00;
        for(int i = 0; i < row_num ; i ++)
        {
            double tmp = lm.Fe_lr(Y[i],H[i]);
            F += tmp;
        }
        
        r_curr = F;
        k++;
    }
    while(!stop(k,r_prev,r_curr,_setting));
    
    delete [] Y;
    delete [] X;
    delete [] H;
    delete [] KKMR;
    
    printf("The final loss: %lf\n",r_curr);
    printf("Number of iteration: %d\n",k);
    printf("Model: ");
    for(int i = 0; i < feature_num; i ++)
    {
        if(i == feature_num - 1)
        {
            printf("%.20f\n",model[i]);
        }
        else
        {
            printf("%.20f, ",model[i]);
        }
        
    }
    
    DM.message("Finish factorize");

}

__host__ bool techniques::stop(int k, double r_prev, double r_curr, setting &setting)
{
    double iter_num = k;
    double difference = abs(r_prev - r_curr);
    
    if( iter_num == setting.iter_num || difference <= setting.error)
    {
        return true;
    }
    else
    {
        return false;
    }
}


/*
 Read a single file the columns of which are in format like: id, label, feature
 The offset entry for W0 is not considered for now
 Logistic Regression for now
 */

//specific techniques selection: flag (for generalization purpose)

__host__ void techniques::SGD(vector< vector<double> > data, setting _setting, double *&model, int feature_num)
{
    DataManagement::message("Start SGD");
    long data_size = data.size();
    vector<long> original_index_set;
    vector<long> shuffling_index;
    //Initialize the original_index_set
    for(long i = 0; i < data_size; i ++)
    {
        original_index_set.push_back(i);
    }
    
    linear_models lm;
    //setting
    double step_size = _setting.step_size;
    
    //Allocate the memory to model
    model = new double[feature_num];
    
    for(int i = 0; i < feature_num; i ++)
    {
        model[i] = 0.00;
        
    }
    
    //Loss Function
    double F = 0.00;
    double r_curr = 0.00;
    double r_prev = 0.00;
    int k = 0;
    
    do
    {
        r_prev = F;
        F = 0.00;
        vector<double> gradient(feature_num,0.00);
        
        //Shuffling
        shuffling_index = shuffle(original_index_set, (unsigned)time(NULL));
        /**
        cout<<"shuffling order:"<<endl;
        for(int i = 0; i < shuffling_index.size(); i ++)
        {
            cout<<shuffling_index.at(i)<<" ";
        }
        cout<<endl;
        **/
        
        for(long j = 0; j < data_size; j ++)
        {
            long cur_index = shuffling_index[j];
            
            //Update the model
            double output = 0.00;
            for(int k = 0; k < feature_num; k ++)
            {
                output += model[k]*data[cur_index][k+2];
            }
            
            for(int k = 0; k < feature_num; k ++)
            {
                gradient[k] = lm.G_lr(data[cur_index][1],output)*data[cur_index][2+k];
                model[k] = model[k]-step_size*gradient[k];
            }
            
        }
        
        //Calculate F
        for(long j = 0; j < data_size; j ++)
        {
            double output = 0.00;
            for(int k = 0; k < feature_num; k ++)
            {
                output += model[k]*data[j][k+2];
            }
            double tmp = lm.Fe_lr(data[j][1], output);
            F += tmp;
        }
        
        r_curr = F;
        k ++;
    }
    while(!stop(k,r_prev,r_curr,_setting));
    
    printf("The final loss: %lf\n",r_curr);
    printf("Number of iteration: %d\n",k);
    printf("Model: ");
    for(int i = 0; i < feature_num; i ++)
    {
        if(i == feature_num - 1)
        {
            printf("%.20f\n",model[i]);
        }
        else
        {
            printf("%.20f, ",model[i]);
        }
        
    }
    
    DataManagement::message("Finish SGD");

}

__host__  void techniques::BGD(vector< vector<double> > data, setting _setting, double *&model, int feature_num)
{
    DataManagement::message("Start BGD");
    long data_size = data.size();
    
    // linear_models lm;
    //setting
    double step_size = _setting.step_size;
    
    //Allocate the memory to the model
    model = new double[feature_num];
    
    for(int i = 0; i < feature_num; i ++)
    {
        model[i] = 0.00;
    }
    
    //Loss Function
    double F = 0.00;
    double r_curr = 0.00;
    double r_prev = 0.00;
    int k = 0;
    
    do
    {
        r_prev = F;
        F = 0.00;
        vector<double> gradient(feature_num,0.00);
        
        for(long j = 0; j < data_size; j ++)
        {
            
            //Update the model
            double output = 0.00;
            for(int k = 0; k < feature_num; k ++)
            {
                output += model[k]*data[j][2+k];
            }
            
            for(int k = 0; k < feature_num; k ++)
            {
                gradient[k] += lm.G_lr(data[j][1],output)*data[j][2+k];
            }
            
        }
        
        
        for(int k = 0; k < feature_num; k ++)
        {
            model[k] = model[k]-step_size*gradient[k];
        }
        
        for(long j = 0; j < data_size; j ++)
        {
            double output = 0.00;
            for(int k = 0; k < feature_num; k ++)
            {
                output += model[k]*data[j][2+k];
            }
            double tmp = lm.Fe_lr(data[j][1], output);
            cout<<"tmp loss: "<<tmp<<endl;
            F += tmp;
        }
        
        
        r_curr = F;
        printf("The loss: %lf\n",F);
        k ++;
    }
    while(!stop(k,r_prev,r_curr,_setting));
    
    printf("The final loss: %lf\n",r_curr);
    printf("Number of iteration: %d\n",k);
    printf("Model: ");
    for(int i = 0; i < feature_num; i ++)
    {
        if(i == feature_num - 1)
        {
            printf("%.20f\n",model[i]);
        }
        else
        {
            printf("%.20f, ",model[i]);
        }
        
    }
    
    DataManagement::message("Finish BGD");

}

__host__ void techniques::classify(vector< vector<double> > data, vector<double> model)
{
    linear_models lm;
    // Count the number of correct classifcation
    long count = 0;
    long data_size =  data.size();
    if(data.at(0).size() != model.size()+2)
    {
        DataManagement::errorMessage("Inconsistent file provided");
    }
    
    int featureNum = (int)model.size();
    for(long i = 0; i < data_size; i ++)
    {
        double actual_label = data[i][1];
        double predicted_label = 0.00;
        double confidence = 0.00;
        double output = 0.00;
        for(int j = 0; j < featureNum; j ++)
        {
            output += model[j]*data[i][2+j];
        }
        cout<<"W^TX: "<<output<<endl;
        confidence = lm.C_lr(output);
        if(confidence > 0.5)
        {
            predicted_label = 1.00;
        }
        else
        {
            predicted_label = -1.00;
        }
        if(actual_label == predicted_label)
        {
            cout<<"Prediction Correct"<<endl;
            count++;
        }
        else
        {
            cout<<"Prediction Wrong"<<endl;
        }
        cout<<"Confidence: "<<confidence<<endl;
        cout<<"Actual Label: "<<actual_label<<","<<"Predicted Label: "<<predicted_label<<endl;
    }
    cout<<"Correcteness: "<<(double)count/(double)data_size<<endl;
    
    cout<<"Predict the newest day: "<<endl;
    vector<double> toBePredicted;
    toBePredicted.push_back(1);
    for(int i = 0; i < 4; i ++)
    {
        toBePredicted.push_back(0);
    }
    
    toBePredicted.push_back(7.93);
    toBePredicted.push_back(8.16);
    toBePredicted.push_back(7.87);
    toBePredicted.push_back(8.06);
    toBePredicted.push_back(23.3105);
    toBePredicted.push_back(8.06);
    toBePredicted.push_back(1);
    toBePredicted.push_back(7.93);
    toBePredicted.push_back(1);
    toBePredicted.push_back(0.15);
    
    double output = 0.00;
    for(int j = 0; j < featureNum; j ++)
    {
        output += model[j]*toBePredicted[j];
    }
    
}

vector<int> techniques::shuffle(vector<int> &index_set, unsigned seed)
{
    vector<int> original_set = index_set;
    int size = (int)index_set.size();
    vector<int> new_index_set;
    srand (seed);
    for(int i = 0; i < size; i ++)
    {
        int cur_size = (int)original_set.size();
        int rand_index = random()%cur_size;
        new_index_set.push_back(original_set.at(rand_index));
        original_set.erase(original_set.begin()+rand_index);
    }
    
    return new_index_set;
}

vector<long> techniques::shuffle(vector<long> &index_set, unsigned seed)
{
    vector<long> original_set = index_set;
    long size = (long)index_set.size();
    vector<long> new_index_set;
    srand(seed);
    for(long i = 0; i < size; i ++)
    {
        long cur_size = original_set.size();
        long rand_index = random()%cur_size;
        new_index_set.push_back(original_set.at(rand_index));
        original_set.erase(original_set.begin()+rand_index);
    }
    
    return new_index_set;
}

// Shuffle mainly focus on file operation, shuffle the index, not much computation, 
// No need for GPU launching
